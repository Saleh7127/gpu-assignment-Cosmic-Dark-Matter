#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>

float *real_rasc, *real_decl, *rand_rasc, *rand_decl;
float pi = 3.14159265;
long int MemoryAllocatedCPU = 0L;

__global__ void Hist(float *real_rasc, float *real_decl, float *rand_rasc, float *rand_decl, unsigned long long int *histogramDR, unsigned long long int *histogramDD, unsigned long long int *histogramRR) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    const float PI = acosf(-1.0f);

    // Ensure that we do not go out of bounds
    if (index < 100000) {
        float real_rasc_val = real_rasc[index];
        float real_decl_val = real_decl[index];
        float tempDR, tempDD, tempRR;
        float angleDR, angleDD, angleRR;

        for (int j = 0; j < 100000; j++) {
            // Compute the cosine of the angle between two sets of coordinates
            tempDR = sinf(real_decl_val) * sinf(rand_decl[j]) + cosf(real_decl_val) * cosf(rand_decl[j]) * cosf(real_rasc_val - rand_rasc[j]);
            angleDR = tempDR >= 1 ? 0 : 180.0f / PI * acosf(tempDR);
            tempDD = sinf(real_decl_val) * sinf(real_decl[j]) + cosf(real_decl_val) * cosf(real_decl[j]) * cosf(real_rasc_val - real_rasc[j]);
            angleDD = tempDD >= 1 ? 0 : 180.0f / PI * acosf(tempDD);
            tempRR = sinf(rand_decl[index]) * sinf(rand_decl[j]) + cosf(rand_decl[index]) * cosf(rand_decl[j]) * cosf(rand_rasc[index] - rand_rasc[j]);
            angleRR = tempRR >= 1 ? 0 : 180.0f / PI * acosf(tempRR);

            // Add the calculated angle to corresponding histogram bin
            atomicAdd(&histogramDR[(int)(angleDR * 4.0f)], 1LL);
            atomicAdd(&histogramDD[(int)(angleDD * 4.0f)], 1LL);
            atomicAdd(&histogramRR[(int)(angleRR * 4.0f)], 1LL);
        }
    }
}

int get_data(int argc, char *argv[]) {
    FILE *real_data_file, *rand_data_file, *out_file;
    float arcmin2rad = 1.0f / 60.0f / 180.0f * pi;
    int Number_of_Galaxies;

    if (argc != 4) {
        printf("   Usage: galaxy real_data flat_data output_file\n   All processes will be killed\n");
        return(1);
    }
    if (argc == 4) {
        printf("   Running galaxy_cuda %s %s %s\n", argv[1], argv[2], argv[3]);

        real_data_file = fopen(argv[1], "r");
        if (real_data_file == NULL) {
            printf("   ERROR: Cannot open real data file %s\n", argv[1]);
            return(1);
        } else {
            fscanf(real_data_file, "%d", &Number_of_Galaxies);
            if (Number_of_Galaxies != 100000) {
                printf("Cannot read file %s correctly, first item not 100000\n", argv[1]);
                fclose(real_data_file);
                return(1);
            }
            for (int i = 0; i < 100000; ++i) {
                float rasc, decl;
                if (fscanf(real_data_file, "%f %f", &rasc, &decl) != 2) {
                    printf("   ERROR: Cannot read line %d in real data file %s\n", i + 1, argv[1]);
                    fclose(real_data_file);
                    return(1);
                }
                real_rasc[i] = rasc * arcmin2rad;
                real_decl[i] = decl * arcmin2rad;
            }
            fclose(real_data_file);
            printf("   Successfully read 100000 lines from %s\n", argv[1]);
        }

        rand_data_file = fopen(argv[2], "r");
        if (rand_data_file == NULL) {
            printf("   ERROR: Cannot open random data file %s\n", argv[2]);
            return(1);
        } else {
            fscanf(rand_data_file, "%d", &Number_of_Galaxies);
            if (Number_of_Galaxies != 100000) {
                printf("Cannot read file %s correctly, first item not 100000\n", argv[2]);
                fclose(rand_data_file);
                return(1);
            }
            for (int i = 0; i < 100000; ++i) {
                float rasc, decl;
                if (fscanf(rand_data_file, "%f %f", &rasc, &decl) != 2) {
                    printf("   ERROR: Cannot read line %d in random data file %s\n", i + 1, argv[2]);
                    fclose(rand_data_file);
                    return(1);
                }
                rand_rasc[i] = rasc * arcmin2rad;
                rand_decl[i] = decl * arcmin2rad;
            }
            fclose(rand_data_file);
            printf("   Successfully read 100000 lines from %s\n", argv[2]);
        }
        
        out_file = fopen(argv[3], "w");
        if (out_file == NULL) {
            printf("   ERROR: Cannot open output file %s\n", argv[3]);
            return(1);
        } else fclose(out_file);
    }

    return(0);
}

int getDevice(void) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("   Found %d CUDA devices\n", deviceCount);
    if (deviceCount < 0 || deviceCount > 128) return (-1);
    
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("      Device %s                  device %d\n", deviceProp.name, device);
        printf("         compute capability           =         %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("         totalGlobalMemory            =        %.2f GB\n", deviceProp.totalGlobalMem / 1000000000.0f);
        printf("         multiProcessorCount          =         %d\n", deviceProp.multiProcessorCount);
        printf("         memoryClockRate              =        %.0f MHz\n", deviceProp.memoryClockRate / 1000.0f);
        printf("         memoryBusWidth               =         %d bits\n", deviceProp.memoryBusWidth);
        printf("         warpSize                     =         %d\n", deviceProp.warpSize);
    }

    hipSetDevice(0);
    hipGetDevice(&device);
    if (device != 0) {
        printf("   Unable to set device 0, using %d instead\n", device);
    } else {
        printf("   Using CUDA device %d\n\n", device);
    }
    return 0;
}

int main(int argc, char* argv[]) {
    int get_data(int argc, char *argv[]);

    hipEvent_t start, stop;
    float elapsedTime;

    // Record start time using CUDA event
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if (getDevice() != 0) return (-1);

    real_rasc = (float *)calloc(100000L, sizeof(float));
    real_decl = (float *)calloc(100000L, sizeof(float));
    rand_rasc = (float *)calloc(100000L, sizeof(float));
    rand_decl = (float *)calloc(100000L, sizeof(float));
    MemoryAllocatedCPU += 10L * 100000L * sizeof(float);
    
    if (get_data(argc, argv) != 0) {
        printf("   Program stopped.\n");
        return (0);
    }
    
    printf("   Input data read, now calculating histograms\n");

    long int histogram_DD[360] = {0L};
    long int histogram_DR[360] = {0L};
    long int histogram_RR[360] = {0L};
    MemoryAllocatedCPU += 3L * 360L * sizeof(long int);

    size_t data_size = 100000 * sizeof(float);
    size_t histo_size = 360 * sizeof(unsigned long long int);
    
    float *real_rasc_gpu; 
    hipMalloc(&real_rasc_gpu, data_size);
    float *real_decl_gpu; 
    hipMalloc(&real_decl_gpu, data_size);
    float *rand_rasc_gpu; 
    hipMalloc(&rand_rasc_gpu, data_size);
    float *rand_decl_gpu; 
    hipMalloc(&rand_decl_gpu, data_size);
    unsigned long long int *histogramDR_gpu; 
    hipMalloc(&histogramDR_gpu, histo_size);
    unsigned long long int *histogramDD_gpu; 
    hipMalloc(&histogramDD_gpu, histo_size);
    unsigned long long int *histogramRR_gpu; 
    hipMalloc(&histogramRR_gpu, histo_size);
    
    hipMemset(histogramDR_gpu, 0, histo_size);
    hipMemset(histogramRR_gpu, 0, histo_size);
    hipMemset(histogramDD_gpu, 0, histo_size);
    
    hipMemcpy(real_rasc_gpu, real_rasc, data_size, hipMemcpyHostToDevice);
    hipMemcpy(real_decl_gpu, real_decl, data_size, hipMemcpyHostToDevice);
    hipMemcpy(rand_rasc_gpu, rand_rasc, data_size, hipMemcpyHostToDevice);
    hipMemcpy(rand_decl_gpu, rand_decl, data_size, hipMemcpyHostToDevice);
    
    int threadsInBlock = 256;
    int blocksInGrid = (100000 + threadsInBlock - 1) / threadsInBlock;

    // Launch the CUDA kernel
    Hist<<<blocksInGrid, threadsInBlock>>>(real_rasc_gpu, real_decl_gpu, rand_rasc_gpu, rand_decl_gpu, histogramDR_gpu, histogramDD_gpu, histogramRR_gpu);

    // Check for any errors in the kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Ensure GPU finishes before measuring end time
    hipDeviceSynchronize();

    // Transfer the computed histograms back to the host
    hipMemcpy(histogram_DD, histogramDD_gpu, histo_size, hipMemcpyDeviceToHost);
    hipMemcpy(histogram_DR, histogramDR_gpu, histo_size, hipMemcpyDeviceToHost);
    hipMemcpy(histogram_RR, histogramRR_gpu, histo_size, hipMemcpyDeviceToHost);

    // Verify the values in the histogram to make sure computations went correctly
    long int histsum = 0L;
    int correct_value = 1;
    
    for (int i = 0; i < 360; ++i) histsum += histogram_DD[i];
    printf("   Histogram DD : sum = %ld\n", histsum);
    if (histsum != 10000000000L) correct_value = 0;

    histsum = 0L;
    for (int i = 0; i < 360; ++i) histsum += histogram_DR[i];
    printf("   Histogram DR : sum = %ld\n", histsum);
    if (histsum != 10000000000L) correct_value = 0;

    histsum = 0L;
    for (int i = 0; i < 360; ++i) histsum += histogram_RR[i];
    printf("   Histogram RR : sum = %ld\n", histsum);
    if (histsum != 10000000000L) correct_value = 0;

    if (correct_value != 1) {
        printf("   Histogram sums should be 10000000000. Ending program prematurely\n");
        return (0);
    }

    // Print some omega values and results
    printf("   Some Omega values for the histograms are given below (For all, please check result file):\n");
    float omega[360];
    for (int i = 0; i < 360; ++i)
        if (histogram_RR[i] != 0L) {
            omega[i] = (histogram_DD[i] - 2L * histogram_DR[i] + histogram_RR[i]) / ((float)(histogram_RR[i]));
            if (i < 5) printf("      Angle %.2f degree => %.2f degree: %.3f\n", i * 0.25f, (i + 1) * 0.25f, omega[i]);
        }

    // Open output file
    FILE *out_file = fopen(argv[3], "w");
    if (out_file == NULL) {
        printf("   ERROR: Cannot open output file %s\n", argv[3]);
    } else {
        // Print the headers for the table
        fprintf(out_file, "Bin\tDR\tDD\tRR\tOmega\n");
        
        // Print the histogram data and Omega values
        for (int i = 0; i < 360; ++i) {
            if (histogram_RR[i] != 0L) { // If RR is non-zero, write data to file
                omega[i] = (histogram_DD[i] - 2L * histogram_DR[i] + histogram_RR[i]) / ((float)(histogram_RR[i]));
                
                // Write the bin (angle), DR, DD, RR, and Omega values to the file
                fprintf(out_file, "%d\t%ld\t%ld\t%ld\t%.6f\n", 
                        i,                             // Bin index
                        histogram_DR[i],                // DR count
                        histogram_DD[i],                // DD count
                        histogram_RR[i],                // RR count
                        omega[i]);                      // Omega value
            }
        }
        fclose(out_file);
    }

    // Record end time and calculate elapsed time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("   Execution time is %.3f milliseconds.\n", elapsedTime);

    return 0;
}
